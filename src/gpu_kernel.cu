#include "gpu_kernel.h"
#include "trace.h"
#include "timer.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiptensor.h>

#include <unordered_map>
#include <vector>
#include <stdio.h>
#include <stdlib.h>

//cTensor Error Handler
#define HANDLE_ERROR(x) { \
  const auto err = x;     \
  if( err != HIPTENSOR_STATUS_SUCCESS ) \
  { \
    printf("Error: %s in line %d\n", hiptensorGetErrorString(err), __LINE__); \
    exit(-1); \
  } \
}


void single_index_contract(std::complex<double> *res, std::complex<double> *bpropMat, std::complex<double> *bsinkMat, long int dim)
{
  //cublas setup - taken from examples online
  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  hipStream_t stream;
  hipStreamCreate(&stream);
  stat = hipblasCreate(&handle);
  hipblasSetStream(handle, stream);
  // cublas multiply can do C=alphaA*B + betaC or something like that
  std::complex<double> alpha(1.,0.);
  std::complex<double> beta(0.,0.);
  hipDoubleComplex *_alpha = reinterpret_cast<hipDoubleComplex*>(&alpha);
  hipDoubleComplex *_beta = reinterpret_cast<hipDoubleComplex*>(&beta);
  int block_size = 32;
  dim3 threads(block_size, block_size);
  dim3 grid(dim/threads.x, dim/threads.y);

  long int bTensor_size = dim*dim*dim*dim*sizeof(std::complex<double>);
  long int resTensor_size = dim*dim*dim*dim*dim*dim*sizeof(std::complex<double>);

  hipDoubleComplex *d_A, *d_B, *d_C;
  hipMalloc((void **) &d_A, bTensor_size);
  hipMalloc((void **) &d_B, bTensor_size);
  hipMalloc((void **) &d_C, resTensor_size);
 
  hipMemcpy(d_A, bpropMat, bTensor_size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, bsinkMat, bTensor_size, hipMemcpyHostToDevice);
 
  Timer<> gpu_timer("Zgemm time");
  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
              //Left index size of A, Right index size of B, summed index
              dim*dim*dim, dim*dim*dim, dim, 
              _alpha, d_A, dim*dim*dim, 
                     d_B, dim, 
              _beta, d_C, dim*dim*dim);
  gpu_timer.stop<std::chrono::microseconds>("us");
 
  hipMemcpy(res, d_C, resTensor_size, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  
  hipblasDestroy(handle);
  hipStreamSynchronize(0);
  hipStreamDestroy(stream);
}


void all_index_contract(std::complex<double> *res, std::complex<double> *bpropMat, std::complex<double> *bsinkMat, long int dim)
{
  Timer<> setup_timer("setup all_index_contract");
  
  //cublas setup - taken from examples online
  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  hipStream_t stream;
  hipStreamCreate(&stream);
  stat = hipblasCreate(&handle);
  hipblasSetStream(handle, stream);
  // cublas multiply can do C=alphaA*B + betaC or something like that
  std::complex<double> alpha(1.,0.);
  std::complex<double> beta(0.,0.);
  hipDoubleComplex *_alpha = reinterpret_cast<hipDoubleComplex*>(&alpha);
  hipDoubleComplex *_beta = reinterpret_cast<hipDoubleComplex*>(&beta);
  int block_size = 32;
  dim3 threads(block_size, block_size);
  dim3 grid(dim/threads.x, dim/threads.y);

  long int bTensor_size = dim*dim*dim*dim*sizeof(std::complex<double>);
  long int resTensor_size = dim*dim*dim*dim*dim*dim*sizeof(std::complex<double>);

  hipDoubleComplex *d_A, *d_B, *d_C, *d_res;
  hipMalloc((void **) &d_A, bTensor_size);
  hipMalloc((void **) &d_B, bTensor_size);
  hipMalloc((void **) &d_C, resTensor_size);
  hipMalloc((void **) &d_res, 2*sizeof(std::complex<double>));
  

  //d_res[0] = make_hipDoubleComplex(0.,0.);
  hipMemcpy(d_A, bpropMat, bTensor_size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, bsinkMat, bTensor_size, hipMemcpyHostToDevice);
  setup_timer.stop<std::chrono::microseconds>("us");

  Timer<> gpu_timer("Zgemm time all");
  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
              //Left index size of A, Right index size of B, summed index
              dim*dim*dim, dim*dim*dim, dim, 
              _alpha, d_A, dim*dim*dim, 
                     d_B, dim, 
              _beta, d_C, dim*dim*dim);
  hipDeviceSynchronize();
  gpu_timer.stop<std::chrono::microseconds>("us");
 
  Timer<> trace_timer("Trace time all");
  trace_rank6<<<1,1>>>(d_res, d_C, dim);
  hipDeviceSynchronize();
  trace_timer.stop<std::chrono::microseconds>("us");

  hipMemcpy(res, d_res, sizeof(std::complex<double>), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipFree(d_res);
  
  hipblasDestroy(handle);
  hipStreamSynchronize(0);
  hipStreamDestroy(stream);
}


void contract(std::complex<double> *res, std::complex<double> *bpropMat, std::complex<double> *bsinkMat, long int dim)
{
  Timer<> setup_timer("setup contract");
  
  //cublas setup - taken from examples online
  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  hipStream_t stream;
  hipStreamCreate(&stream);
  stat = hipblasCreate(&handle);
  hipblasSetStream(handle, stream);
  // cublas multiply can do C=alphaA*B + betaC or something like that
  std::complex<double> alpha(1.,0.);
  std::complex<double> beta(0.,0.);
  hipDoubleComplex *_alpha = reinterpret_cast<hipDoubleComplex*>(&alpha);
  hipDoubleComplex *_beta = reinterpret_cast<hipDoubleComplex*>(&beta);
  int block_size = 32;
  dim3 threads(block_size, block_size);
  dim3 grid(dim/threads.x, dim/threads.y);

  long int bTensor_size = dim*dim*dim*dim*sizeof(std::complex<double>);
  long int resTensor_size = dim*dim*dim*dim*dim*dim*sizeof(std::complex<double>);

  hipDoubleComplex *d_A, *d_B, *d_C, *d_res;
  hipMalloc((void **) &d_A, bTensor_size);
  hipMalloc((void **) &d_B, bTensor_size);
  hipMalloc((void **) &d_C, resTensor_size);
  hipMalloc((void **) &d_res, 2*sizeof(std::complex<double>));
  

  //d_res[0] = make_hipDoubleComplex(0.,0.);
  hipMemcpy(d_A, bpropMat, bTensor_size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, bsinkMat, bTensor_size, hipMemcpyHostToDevice);
  setup_timer.stop<std::chrono::microseconds>("us");
 
  Timer<> contract_timer("contract time");
  contract<<<1,1>>>(d_res, d_A, d_B, dim);
  hipDeviceSynchronize();
  contract_timer.stop<std::chrono::microseconds>("us");

  hipMemcpy(res, d_res, sizeof(std::complex<double>), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipFree(d_res);
  
  hipblasDestroy(handle);
  hipStreamSynchronize(0);
  hipStreamDestroy(stream);
}

void contract4(std::complex<double> *res, std::complex<double> *A, std::complex<double> *B, std::complex<double> *C, std::complex<double> *D, long int dim)
{
  Timer<> setup_timer("setup contract");
  
  //cublas setup - taken from examples online
  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  hipStream_t stream;
  hipStreamCreate(&stream);
  stat = hipblasCreate(&handle);
  hipblasSetStream(handle, stream);
  // cublas multiply can do C=alphaA*B + betaC or something like that
  std::complex<double> alpha(1.,0.);
  std::complex<double> beta(0.,0.);
  hipDoubleComplex *_alpha = reinterpret_cast<hipDoubleComplex*>(&alpha);
  hipDoubleComplex *_beta = reinterpret_cast<hipDoubleComplex*>(&beta);
  int block_size = 32;
  dim3 threads(block_size, block_size);
  dim3 grid(dim/threads.x, dim/threads.y);

  long int bTensor_size = dim*dim*dim*dim*sizeof(std::complex<double>);
  long int resTensor_size = dim*dim*dim*dim*dim*dim*sizeof(std::complex<double>);

  hipDoubleComplex *d_A, *d_B, *d_C, *d_D, *d_AB, *d_CD, *d_res;
  hipMalloc((void **) &d_A, bTensor_size);
  hipMalloc((void **) &d_B, bTensor_size);
  hipMalloc((void **) &d_C, bTensor_size);
  hipMalloc((void **) &d_D, bTensor_size);
  hipMalloc((void **) &d_AB, bTensor_size);
  hipMalloc((void **) &d_CD, bTensor_size);
  hipMalloc((void **) &d_res, 2*sizeof(std::complex<double>));
   

  //d_res[0] = make_hipDoubleComplex(0.,0.);
  hipMemcpy(d_A, A, bTensor_size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, bTensor_size, hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, bTensor_size, hipMemcpyHostToDevice);
  hipMemcpy(d_D, D, bTensor_size, hipMemcpyHostToDevice);
  hipMemset(d_AB, 0, bTensor_size);
  hipMemset(d_CD, 0, bTensor_size);
  hipMemset(d_AB, 0, bTensor_size);
  hipMemset(d_res, 0, 2*sizeof(std::complex<double>));
  setup_timer.stop<std::chrono::microseconds>("us");
 
  Timer<> contract_timer("contract time");
  contractAB<<<1,1>>>(d_AB, d_A, d_B, dim);
  contractCD<<<1,1>>>(d_CD, d_C, d_D, dim);
  contract<<<1,1>>>(d_res, d_AB, d_CD, dim);
  hipDeviceSynchronize();
  contract_timer.stop<std::chrono::microseconds>("us");

  hipMemcpy(res, d_res, sizeof(std::complex<double>), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipFree(d_res);
  
  hipblasDestroy(handle);
  hipStreamSynchronize(0);
  hipStreamDestroy(stream);
}


void cuTensorContract(std::complex<double> *res, std::complex<double> *A,std::complex<double> *B, long int dim)
{
  // allocate device memory and copy tensors
  long int bTensor_size = dim*dim*dim*dim*sizeof(std::complex<double>);

  hipDoubleComplex *d_A, *d_B, *d_C, *d_D;
  hipMalloc((void **) &d_A, bTensor_size);
  hipMalloc((void **) &d_B, bTensor_size);
  hipMalloc((void **) &d_C, dim*dim*sizeof(std::complex<double>));
  hipMalloc((void **) &d_D, sizeof(std::complex<double>));

  hipMemcpy(d_A, A, bTensor_size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, bTensor_size, hipMemcpyHostToDevice);
  
  hipMemset(d_C, 0, dim*dim*sizeof(std::complex<double>));
  hipMemset(d_D, 0, sizeof(std::complex<double>));


  //types of cuTensor
  Timer<> cutensor_setup("cutensor setup time");
  hipDataType tensType = HIP_C_64F;
  hiptensorComputeDescriptor_t computeType = HIPTENSOR_COMPUTE_DESC_64F;

  typedef float floatTypeCompute;

  hipDoubleComplex alpha = make_hipDoubleComplex(1.0,0.0);
  hipDoubleComplex beta = make_hipDoubleComplex(0.0,0.0);

  //modes of tensors
  std::vector<int> modeC{'a','b'};
  std::vector<int> modeA{'a','j','k','l'};
  std::vector<int> modeB{'l','k','j','b'};

  int nmodeA = modeA.size();
  int nmodeB = modeB.size();
  int nmodeC = modeC.size();

  //extents of modes
  std::unordered_map<int, int64_t> extent;
  extent['i']=dim;
  extent['j']=dim;
  extent['k']=dim;
  extent['l']=dim;
  extent['a']=dim;
  extent['b']=dim;

  std::vector<int64_t> extentA, extentB, extentC;
  for(auto mode: modeA)
    extentA.push_back(extent[mode]);
  for(auto mode: modeB)
    extentB.push_back(extent[mode]);
  for(auto mode: modeC)
    extentC.push_back(extent[mode]); 

  //create tensor descriptors 
  hiptensorHandle_t handle;
  cutensorInit(&handle);

  hiptensorTensorDescriptor_t descA, descB, descC;
  HANDLE_ERROR( hiptensorInitTensorDescriptor(
      &handle,
      &descA,
      nmodeA,
      extentA.data(),
      NULL,/*stride*/
      tensType,
      HIPTENSOR_OP_IDENTITY/*applied to each element*/
    )
  );
  HANDLE_ERROR( hiptensorInitTensorDescriptor(
      &handle,
      &descB,
      nmodeB,
      extentB.data(),
      NULL,/*stride*/
      tensType,
      HIPTENSOR_OP_IDENTITY/*applied to each element*/
    )
  );
  HANDLE_ERROR( hiptensorInitTensorDescriptor(
      &handle,
      &descC,
      nmodeC,
      extentC.data(),
      NULL,/*stride*/
      tensType,
      HIPTENSOR_OP_IDENTITY/*applied to each element*/
    )
  );

  //get alignments of A,B,C
  uint32_t alignmentA, alignmentB, alignmentC;
  HANDLE_ERROR( cutensorGetAlignmentRequirement (
    &handle,
    d_A,
    &descA,
    &alignmentA
    )
  );
  HANDLE_ERROR( cutensorGetAlignmentRequirement (
    &handle,
    d_B,
    &descB,
    &alignmentB
    )
  );
  HANDLE_ERROR( cutensorGetAlignmentRequirement (
    &handle,
    d_C,
    &descC,
    &alignmentC
    )
  );

  //create descriptor of contraction
  cutensorContractionDescriptor_t desc;
  HANDLE_ERROR( cutensorInitContractionDescriptor( 
    &handle,
    &desc,
    &descA, modeA.data(), alignmentA,
    &descB, modeB.data(), alignmentB,
    &descC, modeC.data(), alignmentC,
    &descC, modeC.data(), alignmentC,
    computeType
    )
  );


  //determine algorithm
  cutensorContractionFind_t find;
  HANDLE_ERROR( cutensorInitContractionFind(
    &handle, 
    &find,
    HIPTENSOR_ALGO_DEFAULT /*will allow internal heuristic to choose best approach*/    
    )
  );
  
  //query workspace
  size_t worksize = 0;
  HANDLE_ERROR( cutensorContractionGetWorkspace(
    &handle,
    &desc,
    &find,
    CUTENSOR_WORKSPACE_RECOMMENDED,
    &worksize
    )
  );

  //allocate workspace
  void *work = nullptr;
  if(worksize > 0)
  {
    if( hipSuccess != hipMalloc(&work, worksize) )
    {
      work = nullptr;
      worksize=0;
    }
  }

  //create contraction plan
  hiptensorContractionPlan_t plan;
  HANDLE_ERROR( cutensorInitContractionPlan(
    &handle,
    &plan,
    &desc,
    &find,
    worksize
    )
  );

  cutensor_setup.stop<std::chrono::microseconds>("us");
  
  
  hiptensorStatus_t err;
  
  Timer<> cutensor_contract("cutensor contract");
  //EXECUTE IT!
  err = hiptensorContraction(
      &handle, &plan,
      &alpha, d_A, 
                     d_B,
      &beta, d_C,
                    d_C,
      work, worksize,
      0/*stream*/
  );

  hipDeviceSynchronize();

  cutensor_contract.stop<std::chrono::microseconds>("us");
  
  if(err != HIPTENSOR_STATUS_SUCCESS)
  {
    printf("ERROR: %s\n", hiptensorGetErrorString(err));
  }

  trace_matrix<<<1,1>>>(d_D, d_C, dim);

  hipMemcpy(res, d_D, sizeof(std::complex<double>), hipMemcpyDeviceToHost);

  if(d_A)
    hipFree(d_A);
  if(d_B)
    hipFree(d_B);
  if(d_C)
    hipFree(d_C);
  if(d_D)
    hipFree(d_D);
  if(work) 
    hipFree(work);
}
