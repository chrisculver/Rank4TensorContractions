#include "gpu_kernel.h"
#include "trace.h"
#include "timer.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiptensor.h>

#include <unordered_map>
#include <vector>
#include <stdio.h>
#include <stdlib.h>

//cTensor Error Handler
#define HANDLE_ERROR(x) { \
  const auto err = x;     \
  if( err != HIPTENSOR_STATUS_SUCCESS ) \
  { \
    printf("Error: %s in line %d\n", hiptensorGetErrorString(err), __LINE__); \
    exit(-1); \
  } \
}


void single_index_contract(std::complex<double> *res, std::complex<double> *bpropMat, std::complex<double> *bsinkMat, long int dim)
{
  //cublas setup - taken from examples online
  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  hipStream_t stream;
  hipStreamCreate(&stream);
  stat = hipblasCreate(&handle);
  hipblasSetStream(handle, stream);
  // cublas multiply can do C=alphaA*B + betaC or something like that
  std::complex<double> alpha(1.,0.);
  std::complex<double> beta(0.,0.);
  hipDoubleComplex *_alpha = reinterpret_cast<hipDoubleComplex*>(&alpha);
  hipDoubleComplex *_beta = reinterpret_cast<hipDoubleComplex*>(&beta);
  int block_size = 32;
  dim3 threads(block_size, block_size);
  dim3 grid(dim/threads.x, dim/threads.y);

  long int bTensor_size = dim*dim*dim*dim*sizeof(std::complex<double>);
  long int resTensor_size = dim*dim*dim*dim*dim*dim*sizeof(std::complex<double>);

  hipDoubleComplex *d_A, *d_B, *d_C;
  hipMalloc((void **) &d_A, bTensor_size);
  hipMalloc((void **) &d_B, bTensor_size);
  hipMalloc((void **) &d_C, resTensor_size);
 
  hipMemcpy(d_A, bpropMat, bTensor_size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, bsinkMat, bTensor_size, hipMemcpyHostToDevice);
 
  Timer<> gpu_timer("Zgemm time");
  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
              //Left index size of A, Right index size of B, summed index
              dim*dim*dim, dim*dim*dim, dim, 
              _alpha, d_A, dim*dim*dim, 
                     d_B, dim, 
              _beta, d_C, dim*dim*dim);
  gpu_timer.stop<std::chrono::microseconds>("us");
 
  hipMemcpy(res, d_C, resTensor_size, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  
  hipblasDestroy(handle);
  hipStreamSynchronize(0);
  hipStreamDestroy(stream);
}


void all_index_contract(std::complex<double> *res, std::complex<double> *bpropMat, std::complex<double> *bsinkMat, long int dim)
{
  Timer<> setup_timer("setup all_index_contract");
  
  //cublas setup - taken from examples online
  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  hipStream_t stream;
  hipStreamCreate(&stream);
  stat = hipblasCreate(&handle);
  hipblasSetStream(handle, stream);
  // cublas multiply can do C=alphaA*B + betaC or something like that
  std::complex<double> alpha(1.,0.);
  std::complex<double> beta(0.,0.);
  hipDoubleComplex *_alpha = reinterpret_cast<hipDoubleComplex*>(&alpha);
  hipDoubleComplex *_beta = reinterpret_cast<hipDoubleComplex*>(&beta);
  int block_size = 32;
  dim3 threads(block_size, block_size);
  dim3 grid(dim/threads.x, dim/threads.y);

  long int bTensor_size = dim*dim*dim*dim*sizeof(std::complex<double>);
  long int resTensor_size = dim*dim*dim*dim*dim*dim*sizeof(std::complex<double>);

  hipDoubleComplex *d_A, *d_B, *d_C, *d_res;
  hipMalloc((void **) &d_A, bTensor_size);
  hipMalloc((void **) &d_B, bTensor_size);
  hipMalloc((void **) &d_C, resTensor_size);
  hipMalloc((void **) &d_res, 2*sizeof(std::complex<double>));
  

  //d_res[0] = make_hipDoubleComplex(0.,0.);
  hipMemcpy(d_A, bpropMat, bTensor_size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, bsinkMat, bTensor_size, hipMemcpyHostToDevice);
  setup_timer.stop<std::chrono::microseconds>("us");

  Timer<> gpu_timer("Zgemm time all");
  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
              //Left index size of A, Right index size of B, summed index
              dim*dim*dim, dim*dim*dim, dim, 
              _alpha, d_A, dim*dim*dim, 
                     d_B, dim, 
              _beta, d_C, dim*dim*dim);
  hipDeviceSynchronize();
  gpu_timer.stop<std::chrono::microseconds>("us");
 
  Timer<> trace_timer("Trace time all");
  trace_rank6<<<1,1>>>(d_res, d_C, dim);
  hipDeviceSynchronize();
  trace_timer.stop<std::chrono::microseconds>("us");

  hipMemcpy(res, d_res, sizeof(std::complex<double>), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipFree(d_res);
  
  hipblasDestroy(handle);
  hipStreamSynchronize(0);
  hipStreamDestroy(stream);
}


void contract(std::complex<double> *res, std::complex<double> *bpropMat, std::complex<double> *bsinkMat, long int dim)
{
  Timer<> setup_timer("setup contract");
  
  //cublas setup - taken from examples online
  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  hipStream_t stream;
  hipStreamCreate(&stream);
  stat = hipblasCreate(&handle);
  hipblasSetStream(handle, stream);
  // cublas multiply can do C=alphaA*B + betaC or something like that
  std::complex<double> alpha(1.,0.);
  std::complex<double> beta(0.,0.);
  hipDoubleComplex *_alpha = reinterpret_cast<hipDoubleComplex*>(&alpha);
  hipDoubleComplex *_beta = reinterpret_cast<hipDoubleComplex*>(&beta);
  int block_size = 32;
  dim3 threads(block_size, block_size);
  dim3 grid(dim/threads.x, dim/threads.y);

  long int bTensor_size = dim*dim*dim*dim*sizeof(std::complex<double>);
  long int resTensor_size = dim*dim*dim*dim*dim*dim*sizeof(std::complex<double>);

  hipDoubleComplex *d_A, *d_B, *d_C, *d_res;
  hipMalloc((void **) &d_A, bTensor_size);
  hipMalloc((void **) &d_B, bTensor_size);
  hipMalloc((void **) &d_C, resTensor_size);
  hipMalloc((void **) &d_res, 2*sizeof(std::complex<double>));
  

  //d_res[0] = make_hipDoubleComplex(0.,0.);
  hipMemcpy(d_A, bpropMat, bTensor_size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, bsinkMat, bTensor_size, hipMemcpyHostToDevice);
  setup_timer.stop<std::chrono::microseconds>("us");
 
  Timer<> contract_timer("contract time");
  contract<<<1,1>>>(d_res, d_A, d_B, dim);
  hipDeviceSynchronize();
  contract_timer.stop<std::chrono::microseconds>("us");

  hipMemcpy(res, d_res, sizeof(std::complex<double>), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipFree(d_res);
  
  hipblasDestroy(handle);
  hipStreamSynchronize(0);
  hipStreamDestroy(stream);
}

void contract4(std::complex<double> *res, std::complex<double> *A, std::complex<double> *B, std::complex<double> *C, std::complex<double> *D, long int dim)
{
  Timer<> setup_timer("setup contract");
  
  //cublas setup - taken from examples online
  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  hipStream_t stream;
  hipStreamCreate(&stream);
  stat = hipblasCreate(&handle);
  hipblasSetStream(handle, stream);
  // cublas multiply can do C=alphaA*B + betaC or something like that
  std::complex<double> alpha(1.,0.);
  std::complex<double> beta(0.,0.);
  hipDoubleComplex *_alpha = reinterpret_cast<hipDoubleComplex*>(&alpha);
  hipDoubleComplex *_beta = reinterpret_cast<hipDoubleComplex*>(&beta);
  int block_size = 32;
  dim3 threads(block_size, block_size);
  dim3 grid(dim/threads.x, dim/threads.y);

  long int bTensor_size = dim*dim*dim*dim*sizeof(std::complex<double>);
  long int resTensor_size = dim*dim*dim*dim*dim*dim*sizeof(std::complex<double>);

  hipDoubleComplex *d_A, *d_B, *d_C, *d_D, *d_AB, *d_CD, *d_res;
  hipMalloc((void **) &d_A, bTensor_size);
  hipMalloc((void **) &d_B, bTensor_size);
  hipMalloc((void **) &d_C, bTensor_size);
  hipMalloc((void **) &d_D, bTensor_size);
  hipMalloc((void **) &d_AB, bTensor_size);
  hipMalloc((void **) &d_CD, bTensor_size);
  hipMalloc((void **) &d_res, 2*sizeof(std::complex<double>));
   

  //d_res[0] = make_hipDoubleComplex(0.,0.);
  hipMemcpy(d_A, A, bTensor_size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, bTensor_size, hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, bTensor_size, hipMemcpyHostToDevice);
  hipMemcpy(d_D, D, bTensor_size, hipMemcpyHostToDevice);
  hipMemset(d_AB, 0, bTensor_size);
  hipMemset(d_CD, 0, bTensor_size);
  hipMemset(d_AB, 0, bTensor_size);
  hipMemset(d_res, 0, 2*sizeof(std::complex<double>));
  setup_timer.stop<std::chrono::microseconds>("us");
 
  Timer<> contract_timer("contract time");
  contractAB<<<1,1>>>(d_AB, d_A, d_B, dim);
  contractCD<<<1,1>>>(d_CD, d_C, d_D, dim);
  contract<<<1,1>>>(d_res, d_AB, d_CD, dim);
  hipDeviceSynchronize();
  contract_timer.stop<std::chrono::microseconds>("us");

  hipMemcpy(res, d_res, sizeof(std::complex<double>), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipFree(d_res);
  
  hipblasDestroy(handle);
  hipStreamSynchronize(0);
  hipStreamDestroy(stream);
}

