#include "gpu_kernel.h"
#include "trace.h"
#include "timer.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

void single_index_contract(std::complex<double> *res, std::complex<double> *bpropMat, std::complex<double> *bsinkMat, long int dim)
{
  //cublas setup - taken from examples online
  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  hipStream_t stream;
  hipStreamCreate(&stream);
  stat = hipblasCreate(&handle);
  hipblasSetStream(handle, stream);
  // cublas multiply can do C=alphaA*B + betaC or something like that
  std::complex<double> alpha(1.,0.);
  std::complex<double> beta(0.,0.);
  hipDoubleComplex *_alpha = reinterpret_cast<hipDoubleComplex*>(&alpha);
  hipDoubleComplex *_beta = reinterpret_cast<hipDoubleComplex*>(&beta);
  int block_size = 32;
  dim3 threads(block_size, block_size);
  dim3 grid(dim/threads.x, dim/threads.y);

  long int bTensor_size = dim*dim*dim*dim*sizeof(std::complex<double>);
  long int resTensor_size = dim*dim*dim*dim*dim*dim*sizeof(std::complex<double>);

  hipDoubleComplex *d_A, *d_B, *d_C;
  hipMalloc((void **) &d_A, bTensor_size);
  hipMalloc((void **) &d_B, bTensor_size);
  hipMalloc((void **) &d_C, resTensor_size);
 
  hipMemcpy(d_A, bpropMat, bTensor_size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, bsinkMat, bTensor_size, hipMemcpyHostToDevice);
 
  Timer<> gpu_timer("Zgemm time");
  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
              //Left index size of A, Right index size of B, summed index
              dim*dim*dim, dim*dim*dim, dim, 
              _alpha, d_A, dim*dim*dim, 
                     d_B, dim, 
              _beta, d_C, dim*dim*dim);
  gpu_timer.print<std::chrono::microseconds>("us");
 
  hipMemcpy(res, d_C, resTensor_size, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  
  hipblasDestroy(handle);
  hipStreamSynchronize(0);
  hipStreamDestroy(stream);
}


void all_index_contract(std::complex<double> *res, std::complex<double> *bpropMat, std::complex<double> *bsinkMat, long int dim)
{
  Timer<> setup_timer("setup all_index_contract");
  
  //cublas setup - taken from examples online
  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  hipStream_t stream;
  hipStreamCreate(&stream);
  stat = hipblasCreate(&handle);
  hipblasSetStream(handle, stream);
  // cublas multiply can do C=alphaA*B + betaC or something like that
  std::complex<double> alpha(1.,0.);
  std::complex<double> beta(0.,0.);
  hipDoubleComplex *_alpha = reinterpret_cast<hipDoubleComplex*>(&alpha);
  hipDoubleComplex *_beta = reinterpret_cast<hipDoubleComplex*>(&beta);
  int block_size = 32;
  dim3 threads(block_size, block_size);
  dim3 grid(dim/threads.x, dim/threads.y);

  long int bTensor_size = dim*dim*dim*dim*sizeof(std::complex<double>);
  long int resTensor_size = dim*dim*dim*dim*dim*dim*sizeof(std::complex<double>);

  hipDoubleComplex *d_A, *d_B, *d_C, *d_res;
  hipMalloc((void **) &d_A, bTensor_size);
  hipMalloc((void **) &d_B, bTensor_size);
  hipMalloc((void **) &d_C, resTensor_size);
  hipMalloc((void **) &d_res, 2*sizeof(std::complex<double>));
  

  //d_res[0] = make_hipDoubleComplex(0.,0.);
  hipMemcpy(d_A, bpropMat, bTensor_size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, bsinkMat, bTensor_size, hipMemcpyHostToDevice);
  setup_timer.print<std::chrono::microseconds>("us");

  Timer<> gpu_timer("Zgemm time all");
  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
              //Left index size of A, Right index size of B, summed index
              dim*dim*dim, dim*dim*dim, dim, 
              _alpha, d_A, dim*dim*dim, 
                     d_B, dim, 
              _beta, d_C, dim*dim*dim);
  hipDeviceSynchronize();
  gpu_timer.print<std::chrono::microseconds>("us");
 
  Timer<> trace_timer("Trace time all");
  trace_rank6<<<1,1>>>(d_res, d_C, dim);
  hipDeviceSynchronize();
  trace_timer.print<std::chrono::microseconds>("us");

  hipMemcpy(res, d_res, sizeof(std::complex<double>), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipFree(d_res);
  
  hipblasDestroy(handle);
  hipStreamSynchronize(0);
  hipStreamDestroy(stream);
}


void contract(std::complex<double> *res, std::complex<double> *bpropMat, std::complex<double> *bsinkMat, long int dim)
{
  Timer<> setup_timer("setup contract");
  
  //cublas setup - taken from examples online
  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  hipStream_t stream;
  hipStreamCreate(&stream);
  stat = hipblasCreate(&handle);
  hipblasSetStream(handle, stream);
  // cublas multiply can do C=alphaA*B + betaC or something like that
  std::complex<double> alpha(1.,0.);
  std::complex<double> beta(0.,0.);
  hipDoubleComplex *_alpha = reinterpret_cast<hipDoubleComplex*>(&alpha);
  hipDoubleComplex *_beta = reinterpret_cast<hipDoubleComplex*>(&beta);
  int block_size = 32;
  dim3 threads(block_size, block_size);
  dim3 grid(dim/threads.x, dim/threads.y);

  long int bTensor_size = dim*dim*dim*dim*sizeof(std::complex<double>);
  long int resTensor_size = dim*dim*dim*dim*dim*dim*sizeof(std::complex<double>);

  hipDoubleComplex *d_A, *d_B, *d_C, *d_res;
  hipMalloc((void **) &d_A, bTensor_size);
  hipMalloc((void **) &d_B, bTensor_size);
  hipMalloc((void **) &d_C, resTensor_size);
  hipMalloc((void **) &d_res, 2*sizeof(std::complex<double>));
  

  //d_res[0] = make_hipDoubleComplex(0.,0.);
  hipMemcpy(d_A, bpropMat, bTensor_size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, bsinkMat, bTensor_size, hipMemcpyHostToDevice);
  setup_timer.print<std::chrono::microseconds>("us");
 
  Timer<> contract_timer("contract time");
  contract<<<1,1>>>(d_res, d_A, d_B, dim);
  hipDeviceSynchronize();
  contract_timer.print<std::chrono::microseconds>("us");

  hipMemcpy(res, d_res, sizeof(std::complex<double>), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipFree(d_res);
  
  hipblasDestroy(handle);
  hipStreamSynchronize(0);
  hipStreamDestroy(stream);
}
