#include "hip/hip_runtime.h"
#include "tensor_kernel.h"
#include "trace.h"
#include "timer.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiptensor.h>

#include <unordered_map>
#include <vector>
#include <stdio.h>
#include <stdlib.h>

//cTensor Error Handler
#define HANDLE_ERROR(x) { \
  const auto err = x;     \
  if( err != HIPTENSOR_STATUS_SUCCESS ) \
  { \
    printf("Error: %s in line %d\n", hiptensorGetErrorString(err), __LINE__); \
    exit(-1); \
  } \
}

/* 
   specialization of CUTensor information for BB->BB scattering tensors
   IMPORTANT THINGS
      *Assume every mode has the same dimensions
      *Manages GPU memory for the tensor
      *Computes the cutensor desc 
      *Computes the cutensor alignment

   You can change which modes of the tensor will be contracted with change_contraction_modes()
*/
class CUTensor
{
  public: 
    hiptensorHandle_t &handle;
    hipDoubleComplex *data;
    long int dim;

    std::vector<int> modes;
    std::vector<int64_t> extents;
    hiptensorTensorDescriptor_t desc;
    uint32_t alignment;

    CUTensor(std::complex<double> *host_data, hiptensorHandle_t &h, long int dim, std::vector<int> m):
      handle(h), modes(m)
    {
      long int tensor_size = pow(dim, modes.size())*sizeof(std::complex<double>);
      hipMalloc((void **) &data, tensor_size);
      if(host_data)
        hipMemcpy(data, host_data, tensor_size, hipMemcpyHostToDevice);
      else
        hipMemset(data, 0, tensor_size);

      for(auto m: modes)
        extents.push_back(dim);

      init_descriptor();
      init_alignment();


    }

    ~CUTensor()
    {
      if(data)
        hipFree(data);
    }

    void change_contraction_modes(std::vector<int> m)
    {
      if( m.size() != modes.size() )
      {
        printf("Can't change modes while changing rank of tensor!");
        exit(-1);
      }
      modes=m;
      // I **think** these need to get called to update the descriptor and alignment
      init_descriptor();
      init_alignment();
    }

    void init_descriptor()
    {
      HANDLE_ERROR( hiptensorInitTensorDescriptor(
        &handle,
        &desc,
        modes.size(),
        extents.data(),
        NULL,//stride
        HIP_C_64F,
        HIPTENSOR_OP_IDENTITY//applied to each element
        )
      );
    }

    void init_alignment()
    {
      HANDLE_ERROR( cutensorGetAlignmentRequirement (
        &handle,
        data,
        &desc,
        &alignment
        )
      );
    }
};



void cuTensorContract(std::complex<double> *res, std::complex<double> *A,std::complex<double> *B, long int dim)
{
  // allocate device memory and copy tensors
  hipDoubleComplex *d_D;
  hipMalloc((void **) &d_D, sizeof(std::complex<double>));
  hipMemset(d_D, 0, sizeof(std::complex<double>));


  //types of cuTensor
  Timer<> cutensor_setup("cutensor setup time");
  hipDataType tensType = HIP_C_64F;
  hiptensorComputeDescriptor_t computeType = HIPTENSOR_COMPUTE_DESC_64F;

  typedef float floatTypeCompute;

  hipDoubleComplex alpha = make_hipDoubleComplex(1.0,0.0);
  hipDoubleComplex beta = make_hipDoubleComplex(0.0,0.0);

  //modes of tensors
  std::vector<int> modeC{'a','b'};
  std::vector<int> modeA{'a','j','k','l'};
  std::vector<int> modeB{'l','k','j','b'};


  //create tensor descriptors 
  hiptensorHandle_t handle;
  cutensorInit(&handle);

  CUTensor dA(A, handle, dim, modeA);
  CUTensor dB(B, handle, dim, modeB);
  CUTensor dC(nullptr, handle, dim, modeC);
  

  //create descriptor of contraction
  cutensorContractionDescriptor_t desc;
  HANDLE_ERROR( cutensorInitContractionDescriptor( 
    &handle,
    &desc,
    &(dA.desc), dA.modes.data(), dA.alignment,
    &(dB.desc), dB.modes.data(), dB.alignment,
    &(dC.desc), dC.modes.data(), dC.alignment,
    &(dC.desc), dC.modes.data(), dC.alignment,
    computeType
    )
  );


  //determine algorithm
  cutensorContractionFind_t find;
  HANDLE_ERROR( cutensorInitContractionFind(
    &handle, 
    &find,
    HIPTENSOR_ALGO_DEFAULT /*will allow internal heuristic to choose best approach*/    
    )
  );
  
  //query workspace
  size_t worksize = 0;
  HANDLE_ERROR( cutensorContractionGetWorkspace(
    &handle,
    &desc,
    &find,
    CUTENSOR_WORKSPACE_RECOMMENDED,
    &worksize
    )
  );

  //allocate workspace
  void *work = nullptr;
  if(worksize > 0)
  {
    if( hipSuccess != hipMalloc(&work, worksize) )
    {
      work = nullptr;
      worksize=0;
    }
  }

  //create contraction plan
  hiptensorContractionPlan_t plan;
  HANDLE_ERROR( cutensorInitContractionPlan(
    &handle,
    &plan,
    &desc,
    &find,
    worksize
    )
  );

  cutensor_setup.stop<std::chrono::microseconds>("us");
  
  
  hiptensorStatus_t err;
  
  Timer<> cutensor_contract("cutensor contract");
  //EXECUTE IT!
  err = hiptensorContraction(
      &handle, &plan,
      &alpha, dA.data, 
                     dB.data,
      &beta, dC.data,
                    dC.data,
      work, worksize,
      0/*stream*/
  );

  hipDeviceSynchronize();

  cutensor_contract.stop<std::chrono::microseconds>("us");
  
  if(err != HIPTENSOR_STATUS_SUCCESS)
  {
    printf("ERROR: %s\n", hiptensorGetErrorString(err));
  }

  trace_matrix<<<1,1>>>(d_D, dC.data, dim);

  hipMemcpy(res, d_D, sizeof(std::complex<double>), hipMemcpyDeviceToHost);

  if(d_D)
    hipFree(d_D);
  if(work) 
    hipFree(work);
}



void cuTensorContract4(std::complex<double> *res, std::complex<double> *A,std::complex<double> *B, 
                       std::complex<double> *C, std::complex<double> *D, long int dim)
{
  // allocate device memory and copy tensors
  long int bTensor_size = dim*dim*dim*dim*sizeof(std::complex<double>);

  hipDoubleComplex *d_A, *d_B, *d_C, *d_D, *d_AB, *d_CD, *d_Mat, *d_tr;
  hipMalloc((void **) &d_A, bTensor_size);
  hipMalloc((void **) &d_B, bTensor_size);
  hipMalloc((void **) &d_C, bTensor_size);
  hipMalloc((void **) &d_D, bTensor_size);
  hipMalloc((void **) &d_AB, bTensor_size);
  hipMalloc((void **) &d_CD, bTensor_size);
  hipMalloc((void **) &d_Mat, dim*dim*sizeof(std::complex<double>));
  hipMalloc((void **) &d_tr, sizeof(std::complex<double>));

  hipMemcpy(d_A, A, bTensor_size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, bTensor_size, hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, bTensor_size, hipMemcpyHostToDevice);
  hipMemcpy(d_D, D, bTensor_size, hipMemcpyHostToDevice);
  
  hipMemset(d_AB, 0, bTensor_size);
  hipMemset(d_CD, 0, bTensor_size);
  hipMemset(d_Mat, 0, dim*dim*sizeof(std::complex<double>));
  hipMemset(d_tr, 0, sizeof(std::complex<double>));


  //types of cuTensor
  Timer<> cutensor_setup("cutensor setup time");
  hipDataType tensType = HIP_C_64F;
  hiptensorComputeDescriptor_t computeType = HIPTENSOR_COMPUTE_DESC_64F;

  typedef float floatTypeCompute;

  hipDoubleComplex alpha = make_hipDoubleComplex(1.0,0.0);
  hipDoubleComplex beta = make_hipDoubleComplex(0.0,0.0);

  //modes of tensors
  std::vector<int> modeA{'a','b','i','j'};
  std::vector<int> modeB{'j','i','c','d'};
  std::vector<int> modeC{'a','b','i','j'};
  std::vector<int> modeD{'i','j','c','d'};
  std::vector<int> modeAB{'a','b','c','d'};
  std::vector<int> modeCD{'a','b','c','d'};
  std::vector<int> modeCD2{'d','c','b','e'};
  std::vector<int> modeMat{'a','e'};

  int nmodeA = modeA.size();
  int nmodeB = modeB.size();
  int nmodeC = modeC.size();
  int nmodeD = modeD.size();
  int nmodeAB = modeAB.size();
  int nmodeCD = modeCD.size();
  int nmodeCD2 = modeCD2.size();
  int nmodeMat = modeMat.size();

  //extents of modes
  std::unordered_map<int, int64_t> extent;
  extent['i']=dim;
  extent['j']=dim;
  extent['a']=dim;
  extent['b']=dim;
  extent['c']=dim;
  extent['d']=dim;
  extent['e']=dim;

  std::vector<int64_t> extentA, extentB, extentC, extentD, extentAB, extentCD, extentCD2, extentMat;
  for(auto mode: modeA)
    extentA.push_back(extent[mode]);
  for(auto mode: modeB)
    extentB.push_back(extent[mode]);
  for(auto mode: modeC)
    extentC.push_back(extent[mode]); 
  for(auto mode: modeD)
    extentD.push_back(extent[mode]); 
  for(auto mode: modeAB)
    extentAB.push_back(extent[mode]); 
  for(auto mode: modeCD)
    extentCD.push_back(extent[mode]); 
  for(auto mode: modeCD2)
    extentCD2.push_back(extent[mode]); 
  for(auto mode: modeMat)
    extentMat.push_back(extent[mode]); 

  //create tensor descriptors 
  hiptensorHandle_t handle;
  cutensorInit(&handle);

  hiptensorTensorDescriptor_t descA, descB, descC, descD, descAB, descCD, descCD2, descMat;
  HANDLE_ERROR( hiptensorInitTensorDescriptor(
      &handle,
      &descA,
      nmodeA,
      extentA.data(),
      NULL,/*stride*/
      tensType,
      HIPTENSOR_OP_IDENTITY/*applied to each element*/
    )
  );
  HANDLE_ERROR( hiptensorInitTensorDescriptor(
      &handle,
      &descB,
      nmodeB,
      extentB.data(),
      NULL,/*stride*/
      tensType,
      HIPTENSOR_OP_IDENTITY/*applied to each element*/
    )
  );
  HANDLE_ERROR( hiptensorInitTensorDescriptor(
      &handle,
      &descC,
      nmodeC,
      extentC.data(),
      NULL,/*stride*/
      tensType,
      HIPTENSOR_OP_IDENTITY/*applied to each element*/
    )
  );
  HANDLE_ERROR( hiptensorInitTensorDescriptor(
      &handle,
      &descD,
      nmodeD,
      extentD.data(),
      NULL,/*stride*/
      tensType,
      HIPTENSOR_OP_IDENTITY/*applied to each element*/
    )
  );
  HANDLE_ERROR( hiptensorInitTensorDescriptor(
      &handle,
      &descAB,
      nmodeAB,
      extentAB.data(),
      NULL,/*stride*/
      tensType,
      HIPTENSOR_OP_IDENTITY/*applied to each element*/
    )
  );
  HANDLE_ERROR( hiptensorInitTensorDescriptor(
      &handle,
      &descCD,
      nmodeCD,
      extentCD.data(),
      NULL,/*stride*/
      tensType,
      HIPTENSOR_OP_IDENTITY/*applied to each element*/
    )
  );
  HANDLE_ERROR( hiptensorInitTensorDescriptor(
      &handle,
      &descCD2,
      nmodeCD2,
      extentCD2.data(),
      NULL,/*stride*/
      tensType,
      HIPTENSOR_OP_IDENTITY/*applied to each element*/
    )
  );
  HANDLE_ERROR( hiptensorInitTensorDescriptor(
      &handle,
      &descMat,
      nmodeMat,
      extentMat.data(),
      NULL,/*stride*/
      tensType,
      HIPTENSOR_OP_IDENTITY/*applied to each element*/
    )
  );

  //get alignments of A,B,C
  uint32_t alignmentA, alignmentB, alignmentC, alignmentD, alignmentAB, alignmentCD, alignmentCD2, alignmentMat;
  HANDLE_ERROR( cutensorGetAlignmentRequirement (
    &handle,
    d_A,
    &descA,
    &alignmentA
    )
  );
  HANDLE_ERROR( cutensorGetAlignmentRequirement (
    &handle,
    d_B,
    &descB,
    &alignmentB
    )
  );
  HANDLE_ERROR( cutensorGetAlignmentRequirement (
    &handle,
    d_C,
    &descC,
    &alignmentC
    )
  );
  HANDLE_ERROR( cutensorGetAlignmentRequirement (
    &handle,
    d_D,
    &descD,
    &alignmentD
    )
  );
  HANDLE_ERROR( cutensorGetAlignmentRequirement (
    &handle,
    d_AB,
    &descAB,
    &alignmentAB
    )
  );
  HANDLE_ERROR( cutensorGetAlignmentRequirement (
    &handle,
    d_CD,
    &descCD,
    &alignmentCD
    )
  );
  HANDLE_ERROR( cutensorGetAlignmentRequirement (
    &handle,
    d_CD,
    &descCD2,
    &alignmentCD2
    )
  );
  HANDLE_ERROR( cutensorGetAlignmentRequirement (
    &handle,
    d_Mat,
    &descMat,
    &alignmentMat
    )
  );

  //create descriptor of contraction
  cutensorContractionDescriptor_t CdescAB, CdescCD, CdescMat;
  HANDLE_ERROR( cutensorInitContractionDescriptor( 
    &handle,
    &CdescAB,
    &descA, modeA.data(), alignmentA,
    &descB, modeB.data(), alignmentB,
    &descAB, modeAB.data(), alignmentAB,
    &descAB, modeAB.data(), alignmentAB,
    computeType
    )
  );
  HANDLE_ERROR( cutensorInitContractionDescriptor( 
    &handle,
    &CdescCD,
    &descC, modeC.data(), alignmentC,
    &descD, modeD.data(), alignmentD,
    &descCD, modeCD.data(), alignmentCD,
    &descCD, modeCD.data(), alignmentCD,
    computeType
    )
  );
  HANDLE_ERROR( cutensorInitContractionDescriptor( 
    &handle,
    &CdescMat,
    &descAB, modeAB.data(), alignmentAB,
    &descCD2, modeCD2.data(), alignmentCD2,
    &descMat, modeMat.data(), alignmentMat,
    &descMat, modeMat.data(), alignmentMat,
    computeType
    )
  );


  //determine algorithm
  cutensorContractionFind_t find;
  HANDLE_ERROR( cutensorInitContractionFind(
    &handle, 
    &find,
    HIPTENSOR_ALGO_DEFAULT /*will allow internal heuristic to choose best approach*/    
    )
  );
  
  //query workspace
  size_t worksize = 0;
  HANDLE_ERROR( cutensorContractionGetWorkspace(
    &handle,
    &CdescAB,
    &find,
    CUTENSOR_WORKSPACE_RECOMMENDED,
    &worksize
    )
  );

  //allocate workspace
  void *work = nullptr;
  if(worksize > 0)
  {
    if( hipSuccess != hipMalloc(&work, worksize) )
    {
      work = nullptr;
      worksize=0;
    }
  }

  //create contraction plan
  hiptensorContractionPlan_t planAB, planCD, planMat;
  HANDLE_ERROR( cutensorInitContractionPlan(
    &handle,
    &planAB,
    &CdescAB,
    &find,
    worksize
    )
  );
  HANDLE_ERROR( cutensorInitContractionPlan(
    &handle,
    &planCD,
    &CdescCD,
    &find,
    worksize
    )
  );
  HANDLE_ERROR( cutensorInitContractionPlan(
    &handle,
    &planMat,
    &CdescMat,
    &find,
    worksize
    )
  );

  cutensor_setup.stop<std::chrono::microseconds>("us");
  hiptensorStatus_t err;
  
  Timer<> cutensor_contract("cutensor contract");
  //EXECUTE IT!
  err = hiptensorContraction(
      &handle, &planAB,
      &alpha, d_A, 
                     d_B,
      &beta, d_AB,
                    d_AB,
      work, worksize,
      0/*stream*/
  );
  hipDeviceSynchronize();
  if(err != HIPTENSOR_STATUS_SUCCESS)
  {
    printf("ERROR: %s\n", hiptensorGetErrorString(err));
  }
  err = hiptensorContraction(
      &handle, &planCD,
      &alpha, d_C, 
                     d_D,
      &beta, d_CD,
                    d_CD,
      work, worksize,
      0/*stream*/
  );
  hipDeviceSynchronize();
  if(err != HIPTENSOR_STATUS_SUCCESS)
  {
    printf("ERROR: %s\n", hiptensorGetErrorString(err));
  }
  err = hiptensorContraction(
      &handle, &planMat,
      &alpha, d_AB, 
                     d_CD,
      &beta, d_Mat,
                    d_Mat,
      work, worksize,
      0/*stream*/
  );
  hipDeviceSynchronize();
  if(err != HIPTENSOR_STATUS_SUCCESS)
  {
    printf("ERROR: %s\n", hiptensorGetErrorString(err));
  }

  cutensor_contract.stop<std::chrono::microseconds>("us");
  

  trace_matrix<<<1,1>>>(d_tr, d_Mat, dim);

  hipMemcpy(res, d_tr, sizeof(std::complex<double>), hipMemcpyDeviceToHost);

  if(d_A)
    hipFree(d_A);
  if(d_B)
    hipFree(d_B);
  if(d_C)
    hipFree(d_C);
  if(d_D)
    hipFree(d_D);
  if(d_AB)
    hipFree(d_AB);
  if(d_CD)
    hipFree(d_CD);
  if(d_Mat)
    hipFree(d_Mat);
  if(d_tr)
    hipFree(d_tr);
  if(work) 
    hipFree(work);
}
